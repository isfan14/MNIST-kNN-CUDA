#include <stdio.h>
#include <stdlib.h>

#include "helper.cuh"

void cudaCheck(hipError_t error_code, const char *file, int line)
{
  if (error_code != hipSuccess)
  {
    std::cerr << "Cuda Error " << error_code << ": '" << hipGetErrorString(error_code) << "' In file '" << file << "' on line " << line << std::endl;
    // fprintf(stderr, "CUDA Error %d: '%s'. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);

    fflush(stderr);
    exit(error_code);
  }
}
