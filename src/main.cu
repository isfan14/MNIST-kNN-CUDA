#include "hip/hip_runtime.h"
#include "mnist.hpp"

__global__ void GetPixelDistance(int *train_images_pixels, int *test_images_pixels, float *distances, int &test_index)
{
  __shared__ float sum;
  sum = 0;
  __syncthreads();

  int train_index = blockIdx.x;
  int pixel_index_x = threadIdx.x;
  int pixel_index_y = threadIdx.y;

  int train_pixel_index = train_index * IMAGE_W * IMAGE_L + pixel_index_x * IMAGE_W + pixel_index_y;
  int test_pixel_index = test_index * IMAGE_W * IMAGE_L + pixel_index_x * IMAGE_W + pixel_index_y;
  int test_distance_index = test_index * N_TRAIN + train_index;

  int trainPixel = train_images_pixels[train_pixel_index];
  int testPixel = test_images_pixels[test_pixel_index];

  int diff = trainPixel - testPixel;

  sum += diff * diff;
  __syncthreads();

  distances[test_distance_index] = sqrt(sum);
}

int main(int argc, char *argv[])
{
  int *h_train_images_pixels, *h_test_images_pixels, *d_train_images_pixels, *d_test_images_pixels;
  char *h_train_labels, *h_test_labels, *d_train_labels, *d_test_labels;
  float *h_test_distances, *d_test_distances;

  hipHostMalloc(&h_train_images_pixels, N_TRAIN * IMAGE_L * IMAGE_W * sizeof(int));
  hipHostMalloc(&h_test_images_pixels, N_TEST * IMAGE_L * IMAGE_W * sizeof(int));
  hipHostMalloc(&h_train_labels, N_TRAIN * sizeof(char));
  hipHostMalloc(&h_test_labels, N_TEST * sizeof(char));
  hipHostMalloc(&h_test_distances, N_TEST * N_TRAIN * sizeof(float));

  hipMalloc(&d_train_images_pixels, N_TRAIN * IMAGE_L * IMAGE_W * sizeof(int));
  hipMalloc(&d_test_images_pixels, N_TEST * IMAGE_L * IMAGE_W * sizeof(int));
  hipMalloc(&d_train_labels, N_TRAIN * sizeof(char));
  hipMalloc(&d_test_labels, N_TEST * sizeof(char));
  hipMalloc(&d_test_distances, N_TEST * N_TRAIN * sizeof(float));

  std::cout << "reading train data..." << std::endl;
  read_mnist("mnist/train-images.idx3-ubyte", "mnist/train-labels.idx1-ubyte", h_train_images_pixels, h_train_labels);
  std::cout << std::endl;

  std::cout << "reading test data..." << std::endl;
  read_mnist("mnist/t10k-images.idx3-ubyte", "mnist/t10k-labels.idx1-ubyte", h_train_images_pixels, h_test_labels);

  hipMemcpy(d_train_images_pixels, d_train_images_pixels, sizeof(*d_train_images_pixels), hipMemcpyHostToDevice);
  hipMemcpy(d_train_labels, d_train_labels, sizeof(*d_train_labels), hipMemcpyHostToDevice);

  hipMemcpy(d_test_images_pixels, d_test_images_pixels, sizeof(*d_test_images_pixels), hipMemcpyHostToDevice);
  hipMemcpy(d_test_labels, d_test_labels, sizeof(*d_test_labels), hipMemcpyHostToDevice);

  int numBlocks = N_TRAIN;
  dim3 threadsPerBlock(IMAGE_L, IMAGE_W);

  for (int i = 0; i < N_TEST; i++)
  {
    GetPixelDistance<<<numBlocks, threadsPerBlock>>>(d_train_images_pixels, d_test_images_pixels, d_test_distances, i);
  }

  hipMemcpy(h_test_distances, d_test_distances, sizeof(d_test_distances), hipMemcpyDeviceToHost);

  hipHostFree(&h_train_images_pixels);
  hipHostFree(&h_test_images_pixels);
  hipHostFree(&h_train_labels);
  hipHostFree(&h_test_labels);
  hipHostFree(&h_test_distances);

  hipFree(&d_train_images_pixels);
  hipFree(&d_test_images_pixels);
  hipFree(&d_train_labels);
  hipFree(&d_test_labels);
  hipFree(&d_test_distances);
}
