#include "hip/hip_runtime.h"
#include <cmath>

#include "mnist.hpp"
#include "helper.cu"

__global__ void compute_diff(int *train_images_pixels, int *test_images_pixels, float *diffs, int test_index)
{
  int train_index = blockIdx.x;
  int pixel_index_x = threadIdx.x;
  int pixel_index_y = threadIdx.y;

  int train_pixel_index = train_index * IMAGE_SIZE + pixel_index_x * IMAGE_W + pixel_index_y;
  int test_pixel_index = test_index * IMAGE_SIZE + pixel_index_x * IMAGE_W + pixel_index_y;
  int diff_index = train_index * IMAGE_SIZE + pixel_index_x * IMAGE_W + pixel_index_y;

  int trainPixel = train_images_pixels[train_pixel_index];
  int testPixel = test_images_pixels[test_pixel_index];

  float diff = pow((float)(trainPixel - testPixel) / 256, 2);
  diffs[diff_index] = diff;
}

__global__ void compute_distance(float *diffs, float *distances, int test_index)
{
  int train_index = blockIdx.x;
  float sum = 0.0f;

  for (int pixel_index = 0; pixel_index < IMAGE_SIZE; pixel_index++)
  {
    sum += diffs[train_index * IMAGE_SIZE + pixel_index];
  }

  distances[train_index] = sqrt(sum);
}

int main(int argc, char *argv[])
{
  const unsigned long TRAIN_IMAGES_SIZE = TRAIN_SIZE * IMAGE_SIZE * sizeof(int);
  const unsigned long TRAIN_LABELS_SIZE = TRAIN_SIZE * sizeof(int);

  const unsigned long TEST_IMAGES_SIZE = TEST_SIZE * IMAGE_SIZE * sizeof(int);
  const unsigned long TEST_LABELS_SIZE = TEST_SIZE * sizeof(int);

  const unsigned long DIFFS_SIZE = TRAIN_SIZE * IMAGE_SIZE * sizeof(float);
  const unsigned long DISTANCES_SIZE = TRAIN_SIZE * sizeof(float);

  int *h_train_images_pixels, *h_test_images_pixels, *d_train_images_pixels, *d_test_images_pixels;
  int *h_train_labels, *h_test_labels;

  // allocate host memory to hold mnist data
  hipHostMalloc((void **)&h_train_images_pixels, TRAIN_IMAGES_SIZE);
  hipHostMalloc((void **)&h_train_labels, TRAIN_LABELS_SIZE);
  hipHostMalloc((void **)&h_test_images_pixels, TEST_IMAGES_SIZE);
  hipHostMalloc((void **)&h_test_labels, TEST_LABELS_SIZE);

  // load mnist train data to host
  std::cout << "loading train data..." << std::endl;
  read_mnist("mnist/train-images.idx3-ubyte", "mnist/train-labels.idx1-ubyte", h_train_images_pixels, h_train_labels);
  std::cout << std::endl;

  // load mnist test data to host
  std::cout << "loading test data..." << std::endl;
  read_mnist("mnist/t10k-images.idx3-ubyte", "mnist/t10k-labels.idx1-ubyte", h_test_images_pixels, h_test_labels);

  // allocate device memory to hold mnist data
  hipMalloc((void **)&d_train_images_pixels, TRAIN_IMAGES_SIZE);
  hipMalloc((void **)&d_test_images_pixels, TEST_IMAGES_SIZE);

  // copy loaded host mnist data to device memory
  hipMemcpy(d_train_images_pixels, h_train_images_pixels, TRAIN_IMAGES_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_test_images_pixels, h_test_images_pixels, TEST_IMAGES_SIZE, hipMemcpyHostToDevice);

  // free mnist data from host memory
  hipHostFree(h_train_images_pixels);
  hipHostFree(h_test_images_pixels);

  unsigned int true_prediction = 0;
  // define number of blocks
  // each block handle 1 train image
  int numBlocks = TRAIN_SIZE;

  // define number of threads per block
  // using 2d threads correspond to each pixel in a train image
  // each thread handle 1 pixel of distance calculation
  dim3 threadsPerBlock = dim3(IMAGE_L, IMAGE_W);

  // loop through all test data to calculate the distance
  for (int test_index = 0; test_index < TEST_SIZE; test_index++)
  {
    float *d_diffs;
    float *d_distances, *h_distances;

    hipMalloc((void **)&d_diffs, DIFFS_SIZE);

    compute_diff<<<numBlocks, threadsPerBlock>>>(d_train_images_pixels, d_test_images_pixels, d_diffs, test_index);
    CUDACHECK(hipPeekAtLastError());

    hipDeviceSynchronize();
    CUDACHECK(hipPeekAtLastError());

    hipMalloc((void **)&d_distances, DISTANCES_SIZE);

    compute_distance<<<numBlocks, 1>>>(d_diffs, d_distances, test_index);
    CUDACHECK(hipPeekAtLastError());

    hipDeviceSynchronize();
    CUDACHECK(hipPeekAtLastError());

    hipHostMalloc((void **)&h_distances, DISTANCES_SIZE);
    hipMemcpy(h_distances, d_distances, DISTANCES_SIZE, hipMemcpyDeviceToHost);

    hipFree(d_diffs);
    hipFree(d_distances);

    unsigned int best_index = 0;
    float best_distance = h_distances[best_index];

    for (unsigned int j = 1; j < TRAIN_SIZE; j++)
    {
      float distance = h_distances[j];

      if (distance < best_distance)
      {
        best_index = j;
        best_distance = distance;
      }
    }

    int label = h_test_labels[test_index];
    int prediction = h_train_labels[best_index];

    if (label == prediction)
    {
      true_prediction++;
    }

    // std::cout << "i: " << test_index << " label: " << label << " prediction: " << prediction << " distance: " << best_distance << std::endl;

    hipHostFree(h_distances);
  }

  std::cout << "true predictions: " << true_prediction << " percentage: " << true_prediction / TEST_SIZE * 100 << std::endl;

  // free mnist data from device memory
  hipFree(d_train_images_pixels);
  hipFree(d_test_images_pixels);

  // free distances data from host memory
  hipHostFree(h_train_labels);
  hipHostFree(h_test_labels);
}
