#include "hip/hip_runtime.h"
#include <cmath>

#include "../utils/mnist.hpp"
#include "../utils/helper.cuh"

__global__ void compute_diff(int *train_images, int *test_images, float *diffs, size_t test_offset)
{
  int train_idx = blockIdx.x;
  int test_idx = blockIdx.y;
  int pixel_idx = threadIdx.x;

  int train_pixel_idx = train_idx * IMAGE_SIZE + pixel_idx;
  int test_pixel_idx = (test_offset + test_idx) * IMAGE_SIZE + pixel_idx;

  int diff_idx = test_idx * TRAIN_SIZE * IMAGE_SIZE + train_idx * IMAGE_SIZE + pixel_idx;

  diffs[diff_idx] = pow((train_images[train_pixel_idx] - test_images[test_pixel_idx]) / 256.0, 2);
}

__global__ void compute_distance(float *diffs, float *distances)
{
  int train_idx = blockIdx.x;
  int test_idx = blockIdx.y;

  float sum = 0.0;

  for (int pixel_idx = 0; pixel_idx < IMAGE_SIZE; pixel_idx++)
  {
    sum += diffs[test_idx * TRAIN_SIZE * IMAGE_SIZE + train_idx * IMAGE_SIZE + pixel_idx];
  }

  distances[test_idx * TRAIN_SIZE + train_idx] = sqrt(sum);
}

__global__ void predict(float *distances, size_t *best_idxs)
{
  size_t test_idx = blockIdx.y;
  size_t tmp_best_idx = 0;

  float best_distance = distances[test_idx * TRAIN_SIZE + tmp_best_idx];

  for (size_t i = 1; i < TRAIN_SIZE; i++)
  {
    if (distances[test_idx * TRAIN_SIZE + i] < best_distance)
    {
      tmp_best_idx = i;
      best_distance = distances[test_idx * TRAIN_SIZE + tmp_best_idx];
    }
  }

  best_idxs[test_idx] = tmp_best_idx;
}

const size_t PREDICTION_BATCH_SIZE = 40;

const size_t TRAIN_IMAGES_SIZE = TRAIN_SIZE * IMAGE_SIZE * sizeof(int);
const size_t TRAIN_LABELS_SIZE = TRAIN_SIZE * sizeof(int);

const size_t TEST_IMAGES_SIZE = TEST_SIZE * IMAGE_SIZE * sizeof(int);
const size_t TEST_LABELS_SIZE = TEST_SIZE * sizeof(int);

const size_t DIFFS_SIZE = TRAIN_SIZE * IMAGE_SIZE * sizeof(float);
const size_t DISTANCES_SIZE = TRAIN_SIZE * sizeof(float);

int main(int argc, char *argv[])
{
  int *h_train_images_pixels, *h_test_images_pixels, *d_train_images_pixels, *d_test_images_pixels;
  int *h_train_labels, *h_test_labels;

  // allocate host memory to hold mnist data
  hipHostMalloc((void **)&h_train_images_pixels, TRAIN_IMAGES_SIZE);
  hipHostMalloc((void **)&h_train_labels, TRAIN_LABELS_SIZE);
  hipHostMalloc((void **)&h_test_images_pixels, TEST_IMAGES_SIZE);
  hipHostMalloc((void **)&h_test_labels, TEST_LABELS_SIZE);

  // load mnist train data to host
  std::cout << "loading train data..." << std::endl;
  read_mnist("mnist/train-images.idx3-ubyte", "mnist/train-labels.idx1-ubyte", h_train_images_pixels, h_train_labels);
  std::cout << std::endl;

  // load mnist test data to host
  std::cout << "loading test data..." << std::endl;
  read_mnist("mnist/t10k-images.idx3-ubyte", "mnist/t10k-labels.idx1-ubyte", h_test_images_pixels, h_test_labels);
  std::cout << std::endl;

  // copy loaded host mnist data to device memory
  std::cout << "loading train data to device memory..." << std::endl;
  hipMalloc((void **)&d_train_images_pixels, TRAIN_IMAGES_SIZE);
  hipMemcpy(d_train_images_pixels, h_train_images_pixels, TRAIN_IMAGES_SIZE, hipMemcpyHostToDevice);
  hipHostFree(h_train_images_pixels);
  std::cout << std::endl;

  std::cout << "loading test data to device memory..." << std::endl;
  hipMalloc((void **)&d_test_images_pixels, TEST_IMAGES_SIZE);
  hipMemcpy(d_test_images_pixels, h_test_images_pixels, TEST_IMAGES_SIZE, hipMemcpyHostToDevice);
  hipHostFree(h_test_images_pixels);
  std::cout << std::endl;

  float *d_diffs, *d_distances;

  size_t *d_best_idxs, *h_best_idxs;

  int label;
  int prediction;
  size_t true_predictions = 0;

  // some events to count the execution time
  hipEvent_t start_event, diff_start, distance_start, predict_start, end_event;

  hipEventCreate(&start_event);
  hipEventCreate(&diff_start);
  hipEventCreate(&distance_start);
  hipEventCreate(&predict_start);
  hipEventCreate(&end_event);

  float elapsed_time_ms, total_time_ms, grand_total_time_ms;

  hipMalloc((void **)&d_diffs, PREDICTION_BATCH_SIZE * DIFFS_SIZE);
  hipMalloc((void **)&d_distances, PREDICTION_BATCH_SIZE * DISTANCES_SIZE);
  hipMalloc((void **)&d_best_idxs, PREDICTION_BATCH_SIZE * sizeof(size_t));
  hipHostMalloc((void **)&h_best_idxs, PREDICTION_BATCH_SIZE * sizeof(size_t));

  // int batch = 0;
  hipEventRecord(start_event, 0);
  for (size_t test_offset = 0; test_offset < TEST_SIZE; test_offset += PREDICTION_BATCH_SIZE)
  {
    // std::cout << "batch: " << batch++ << " offset: " << test_offset << std::endl;
    // 1. compute euclidean distance (distance = sqrt(diff))
    // 1.1. compute diff (inside of sqrt)
    // std::cout << "computing diffs : ";
    // hipEventRecord(diff_start, 0);
    compute_diff<<<dim3(TRAIN_SIZE, PREDICTION_BATCH_SIZE), IMAGE_SIZE>>>(d_train_images_pixels, d_test_images_pixels, d_diffs, test_offset);
    CUDACHECK(hipPeekAtLastError());

    // hipEventRecord(distance_start, 0);
    // hipEventSynchronize(distance_start);
    // hipEventElapsedTime(&elapsed_time_ms, diff_start, distance_start);
    // std::cout << elapsed_time_ms << "ms" << std::endl;

    // 1.2. compute distance
    // std::cout << "computing dist. : ";
    compute_distance<<<dim3(TRAIN_SIZE, PREDICTION_BATCH_SIZE), 1>>>(d_diffs, d_distances);
    CUDACHECK(hipPeekAtLastError());

    // hipEventRecord(predict_start, 0);
    // hipEventSynchronize(predict_start);
    // hipEventElapsedTime(&elapsed_time_ms, distance_start, predict_start);
    // std::cout << elapsed_time_ms << "ms" << std::endl;

    // 2. find the closest train image to the current test image
    hipMemset(d_best_idxs, 0, PREDICTION_BATCH_SIZE * sizeof(size_t));
    // std::cout << "predicting      : ";
    predict<<<dim3(1, PREDICTION_BATCH_SIZE), 1>>>(d_distances, d_best_idxs);
    CUDACHECK(hipPeekAtLastError());

    // hipEventRecord(end_event, 0);
    // hipEventSynchronize(end_event);
    // hipEventElapsedTime(&elapsed_time_ms, predict_start, end_event);
    // std::cout << elapsed_time_ms << "ms" << std::endl;
    // std::cout << std::endl;

    hipMemcpy(h_best_idxs, d_best_idxs, PREDICTION_BATCH_SIZE * sizeof(size_t), hipMemcpyDeviceToHost);

    for (size_t test_idx = 0; test_idx < PREDICTION_BATCH_SIZE; test_idx++)
    {
      label = h_test_labels[test_offset + test_idx];
      prediction = h_train_labels[h_best_idxs[test_idx]];
      // std::cout << test_idx << ": label: " << label << " best_idx: " << h_best_idxs[test_idx] << std::endl;
      if (label == prediction)
      {
        true_predictions++;
      }
    }
  }
  hipEventRecord(end_event, 0);
  hipEventSynchronize(end_event);
  hipEventElapsedTime(&elapsed_time_ms, start_event, end_event);

  float percentage = true_predictions * 100.0 / TEST_SIZE;

  std::cout << "true predictions: " << true_predictions << " percentage: " << percentage << " elapsed: " << elapsed_time_ms << " ms" << std::endl;

  hipFree(d_train_images_pixels);
  hipFree(d_test_images_pixels);
  hipFree(d_diffs);
  hipFree(d_distances);
  hipFree(d_best_idxs);

  hipHostFree(h_train_labels);
  hipHostFree(h_test_labels);

  hipHostFree(h_best_idxs);
}
